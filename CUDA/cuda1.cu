/*
 * Almost the minimal CUDA C++ example.
 *
 * Compile with `nvcc -o cuda1 cuda1.cu`
 */


#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

/*
 * Square all elements of array
 */
static __global__
void square(float *x) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    x[i] = x[i]*x[i];
}

int main(int argc, char **argv) {
    /*
     * From CUDA documentation:
     * "There is no explicit initialization function for the runtime;
     * it initializes the first time a runtime function is called"
     */

    /*
     * Create data on CPU
     */
    float *host_data = new float[256];

    for (int i = 0; i < 256; ++i) {
        host_data[i] = float(i);
    }

    /*
     * Copy data to GPU
     */
    float *device_data;
    hipMalloc(&device_data, 256*sizeof(float));
    hipMemcpy(device_data, host_data, 256*sizeof(float), hipMemcpyHostToDevice);

    /*
     * Call `square` with array.
     * Using 2 blocks, each with 128 threads, to evaluate 256 elements.
     */
    square<<<2, 128>>>(device_data);

    /*
     * Return data to CPU
     */
    hipMemcpy(host_data, device_data, 256*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < 256; ++i) {
        cout << host_data[i] << ' ';
    }
    cout << endl;

    hipFree(device_data);
    free(host_data);
}
